#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "histogram_eq.h"
#include "timer.h"

#define THREADS_PER_BLOCK 256

namespace cp
{
    constexpr auto HISTOGRAM_LENGTH = 256;

    __device__ float prob(const int x, const int size)
    {
        return static_cast<float>(x) / size;
    }

    __device__ unsigned char clamp(unsigned char x)
    {
        return min(max(x, static_cast<unsigned char>(0)), static_cast<unsigned char>(255));
    }

    __device__ unsigned char correct_color(float cdf_val, float cdf_min)
    {
        return clamp(static_cast<unsigned char>(255 * (cdf_val - cdf_min) / (1 - cdf_min)));
    }

    __global__ void process_image_and_build_histogram(const float *input_image_data,
                                                      unsigned char *uchar_image,
                                                      unsigned char *gray_image,
                                                      int *histogram,
                                                      int size,
                                                      int size_channels)
    {
        // histogram shared with all threads, each writes 1 position
        // might need to change strategy to write to original histogram
        extern __shared__ int local_histogram[];

        // Initialize shared histogram
        for (int i = threadIdx.x; i < HISTOGRAM_LENGTH; i += blockDim.x)
        {
            local_histogram[i] = 0;
        }
        __syncthreads();

        // Process image and build local histograms
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
        {
            int idx = 3 * i;
            unsigned char r = static_cast<unsigned char>(255 * input_image_data[idx]);
            unsigned char g = static_cast<unsigned char>(255 * input_image_data[idx + 1]);
            unsigned char b = static_cast<unsigned char>(255 * input_image_data[idx + 2]);

            uchar_image[idx] = r;
            uchar_image[idx + 1] = g;
            uchar_image[idx + 2] = b;

            unsigned char gray = static_cast<unsigned char>(0.21f * r + 0.71f * g + 0.07f * b);
            gray_image[i] = gray;
            atomicAdd(&local_histogram[gray], 1);
        }
        __syncthreads();

        // Merge local histograms into the global histogram
        for (int i = threadIdx.x; i < HISTOGRAM_LENGTH; i += blockDim.x)
        {
            atomicAdd(&histogram[i], local_histogram[i]);
        }
    }

    __global__ void calculate_cdf(const int *histogram, float *cdf, int size)
    {
        __shared__ int temp[HISTOGRAM_LENGTH];
        int idx = threadIdx.x;

        if (idx < HISTOGRAM_LENGTH)
        {
            temp[idx] = histogram[idx];
        }
        __syncthreads();

        if (idx == 0)
        {
            cdf[0] = prob(temp[0], size);
            for (int i = 1; i < HISTOGRAM_LENGTH; i++)
            {
                cdf[i] = cdf[i - 1] + prob(temp[i], size);
            }
        }
    }

    __global__ void find_cdf_min(const float *cdf, float *cdf_min)
    {
        __shared__ float min_val[HISTOGRAM_LENGTH];
        int idx = threadIdx.x;

        if (idx < HISTOGRAM_LENGTH)
        {
            min_val[idx] = cdf[idx];
        }
        __syncthreads();

        for (int stride = HISTOGRAM_LENGTH / 2; stride > 0; stride >>= 1)
        {
            if (idx < stride)
            {
                min_val[idx] = min(min_val[idx], min_val[idx + stride]);
            }
            __syncthreads();
        }

        if (idx == 0)
        {
            *cdf_min = min_val[0];
        }
    }

    __global__ void apply_histogram_equalization_and_convert_to_float(unsigned char *uchar_image, float *output_image_data, const float *cdf, float cdf_min, int size_channels)
    {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size_channels; i += blockDim.x * gridDim.x)
        {
            uchar_image[i] = correct_color(cdf[uchar_image[i]], cdf_min);
            output_image_data[i] = static_cast<float>(uchar_image[i]) / 255.0f;
        }
    }

    void histogram_equalization(const int width, const int height,
                                float *image_data,
                                unsigned char *uchar_image,
                                unsigned char *gray_image,
                                int *histogram,
                                float *cdf)
    {
        const auto size = width * height;
        const auto size_channels = size * 3;
        const auto grid_size = (size_channels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        process_image_and_build_histogram<<<grid_size, THREADS_PER_BLOCK>>>(
            image_data, uchar_image, gray_image, histogram, size, size_channels);

        calculate_cdf<<<1, HISTOGRAM_LENGTH>>>(histogram, cdf, size);

        float *d_cdf_min;
        hipMalloc(&d_cdf_min, sizeof(float));

        // need to correct cdf_min
        find_cdf_min<<<1, HISTOGRAM_LENGTH>>>(cdf, d_cdf_min);

        float h_cdf_min; // Host variable to store cdf_min
        hipMemcpy(&h_cdf_min, d_cdf_min, sizeof(float), hipMemcpyDeviceToHost);

        apply_histogram_equalization_and_convert_to_float<<<grid_size, THREADS_PER_BLOCK>>>(
            uchar_image, image_data, cdf, h_cdf_min, size_channels);
    }

    wbImage_t iterative_histogram_equalization(wbImage_t &input_image, int iterations)
    {
        const auto width = wbImage_getWidth(input_image);
        const auto height = wbImage_getHeight(input_image);
        constexpr auto channels = 3;
        const auto size = width * height;
        const auto size_channels = size * channels;

        wbImage_t output_image = wbImage_new(width, height, channels);
        float *input_image_data = wbImage_getData(input_image);
        float *output_image_data = wbImage_getData(output_image);

        float *d_image;
        unsigned char *d_uchar_image, *d_gray_image;
        int *d_histogram;
        float *d_cdf;

        hipMalloc(&d_image, size_channels * sizeof(float));
        hipMalloc(&d_uchar_image, size_channels * sizeof(unsigned char));
        hipMalloc(&d_gray_image, size * sizeof(unsigned char));
        hipMalloc(&d_histogram, HISTOGRAM_LENGTH * sizeof(int));
        hipMalloc(&d_cdf, HISTOGRAM_LENGTH * sizeof(float));

        hipMemcpy(d_image, input_image_data, size_channels * sizeof(float), hipMemcpyHostToDevice);

        marrow::timer t;
        t.start();

        for (int i = 0; i < iterations; i++)
        {
            histogram_equalization(width, height,
                                   d_image,
                                   d_uchar_image, d_gray_image,
                                   d_histogram, d_cdf);
        }

        t.stop();
        t.output_stats(std::cout);

        hipMemcpy(output_image_data, d_image, size_channels * sizeof(float), hipMemcpyDeviceToHost);

        wbImage_setData(output_image, output_image_data);

        hipFree(d_image);
        hipFree(d_uchar_image);
        hipFree(d_gray_image);
        hipFree(d_histogram);
        hipFree(d_cdf);

        return output_image;
    }
}
